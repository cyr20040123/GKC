#include "hip/hip_runtime.h"
#define _in_
#define _out_

// #define KERNEL_TIME_MEASUREMENT

#define FILTER_KERNEL new_filter2 // modify this to change filter: mm_filter, sign_filter, new_filter, new_filter2
#define STR1(R)  #R
#define STR(R) STR1(R)

#include "gkc_cuda.hpp"
#include "types.h"
#include "utilities.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <string>
#include <thread>
#include <future>
#include <functional>
#include <iostream>

using namespace std;

__device__ __constant__ static const unsigned char d_basemap[256] = {
    255, 255, 255, 255, 255, 255, 255, 255, // 0..7
    255, 255, 255, 255, 255, 255, 255, 255, // 8..15
    255, 255, 255, 255, 255, 255, 255, 255, // 16..23
    255, 255, 255, 255, 255, 255, 255, 255, // 24..31
    255, 255, 255, 255, 255, 255, 255, 255, // 32..39
    255, 255, 255, 255, 255, 255, 255, 255, // 40..47
    255, 255, 255, 255, 255, 255, 255, 255, // 48..55
    255, 255, 255, 255, 255, 255, 255, 255, // 56..63
    255, 0, 255, 1, 255, 255, 255, 2, // 64..71
    255, 255, 255, 255, 255, 255, 255, 255, // 72..79
    255, 255, 255, 255, 3, 0, 255, 255, // 80..87
    255, 255, 255, 255, 255, 255, 255, 255, // 88..95
    255, 0, 255, 1, 255, 255, 255, 2, // 96..103
    255, 255, 255, 255, 255, 255, 255, 255, // 104..111
    255, 255, 255, 255, 3, 0, 255, 255, // 112..119
    255, 255, 255, 255, 255, 255, 255, 255, // 120..127
    255, 255, 255, 255, 255, 255, 255, 255, // 128..135
    255, 255, 255, 255, 255, 255, 255, 255, // 136..143
    255, 255, 255, 255, 255, 255, 255, 255, // 144..151
    255, 255, 255, 255, 255, 255, 255, 255, // 152..159
    255, 255, 255, 255, 255, 255, 255, 255, // 160..167
    255, 255, 255, 255, 255, 255, 255, 255, // 168..175
    255, 255, 255, 255, 255, 255, 255, 255, // 176..183
    255, 255, 255, 255, 255, 255, 255, 255, // 184..191
    255, 255, 255, 255, 255, 255, 255, 255, // 192..199
    255, 255, 255, 255, 255, 255, 255, 255, // 200..207
    255, 255, 255, 255, 255, 255, 255, 255, // 208..215
    255, 255, 255, 255, 255, 255, 255, 255, // 216..223
    255, 255, 255, 255, 255, 255, 255, 255, // 224..231
    255, 255, 255, 255, 255, 255, 255, 255, // 232..239
    255, 255, 255, 255, 255, 255, 255, 255, // 240..247
    255, 255, 255, 255, 255, 255, 255, 255  // 248..255
};

__device__ __constant__ static const unsigned char d_basemap_compl[256] = { // complement base
    255, 255, 255, 255, 255, 255, 255, 255, // 0..7
    255, 255, 255, 255, 255, 255, 255, 255, // 8..15
    255, 255, 255, 255, 255, 255, 255, 255, // 16..23
    255, 255, 255, 255, 255, 255, 255, 255, // 24..31
    255, 255, 255, 255, 255, 255, 255, 255, // 32..39
    255, 255, 255, 255, 255, 255, 255, 255, // 40..47
    255, 255, 255, 255, 255, 255, 255, 255, // 48..55
    255, 255, 255, 255, 255, 255, 255, 255, // 56..63
    255, 3, 255, 2, 255, 255, 255, 1, // 64..71
    255, 255, 255, 255, 255, 255, 255, 255, // 72..79
    255, 255, 255, 255, 0, 3, 255, 255, // 80..87
    255, 255, 255, 255, 255, 255, 255, 255, // 88..95
    255, 3, 255, 2, 255, 255, 255, 1, // 96..103
    255, 255, 255, 255, 255, 255, 255, 255, // 104..111
    255, 255, 255, 255, 0, 3, 255, 255, // 112..119
    255, 255, 255, 255, 255, 255, 255, 255, // 120..127
    255, 255, 255, 255, 255, 255, 255, 255, // 128..135
    255, 255, 255, 255, 255, 255, 255, 255, // 136..143
    255, 255, 255, 255, 255, 255, 255, 255, // 144..151
    255, 255, 255, 255, 255, 255, 255, 255, // 152..159
    255, 255, 255, 255, 255, 255, 255, 255, // 160..167
    255, 255, 255, 255, 255, 255, 255, 255, // 168..175
    255, 255, 255, 255, 255, 255, 255, 255, // 176..183
    255, 255, 255, 255, 255, 255, 255, 255, // 184..191
    255, 255, 255, 255, 255, 255, 255, 255, // 192..199
    255, 255, 255, 255, 255, 255, 255, 255, // 200..207
    255, 255, 255, 255, 255, 255, 255, 255, // 208..215
    255, 255, 255, 255, 255, 255, 255, 255, // 216..223
    255, 255, 255, 255, 255, 255, 255, 255, // 224..231
    255, 255, 255, 255, 255, 255, 255, 255, // 232..239
    255, 255, 255, 255, 255, 255, 255, 255, // 240..247
    255, 255, 255, 255, 255, 255, 255, 255  // 248..255
};

// raw read is not a significant VRAM usage, no need for 2-bit encoding
// the majority VRAM usage is caused by minimizer (positions) etc...

extern Logger *logger;

// =================================================
// ================ CLASS PinnedCSR ================
// =================================================
    PinnedCSR::PinnedCSR(vector<string> &reads) {
        this->n_reads = reads.size();
        size_capacity = 0;
        for (string &read: reads) {
            size_capacity += read.length();
        } // about hipHostAlloc https://zhuanlan.zhihu.com/p/188246455
        CUDA_CHECK(hipHostAlloc((void**)(&reads_offs), (this->n_reads+1)*sizeof(T_CSR_cap), hipHostMallocDefault));
        CUDA_CHECK(hipHostAlloc((void**)(&reads_CSR), size_capacity+1, hipHostMallocDefault));
        char *cur_ptr = reads_CSR;
        T_CSR_cap *offs_ptr = reads_offs;
        *offs_ptr = 0;
        for (string &read: reads) {
            memcpy(cur_ptr, read.c_str(), read.length());
            cur_ptr += read.length();
            offs_ptr++;
            *offs_ptr = *(offs_ptr-1) + read.length();
        }
    }
    PinnedCSR::PinnedCSR(vector<ReadPtr> &reads, bool keep_original/*=true*/) { // for sorting CSR (order the pointers as the sorting result)
        this->n_reads = reads.size();
        size_capacity = 0;
        for (const ReadPtr &read_ptr: reads) {
            size_capacity += read_ptr.len;
        } // about hipHostAlloc https://zhuanlan.zhihu.com/p/188246455
        // cerr<<"Pinned reads = "<<n_reads<<" \ttot_sizes = "<<size_capacity<<" bytes"<<endl;
        CUDA_CHECK(hipHostAlloc((void**)(&reads_offs), (this->n_reads+1)*sizeof(T_CSR_cap), hipHostMallocDefault));
        CUDA_CHECK(hipHostAlloc((void**)(&reads_CSR), size_capacity+1, hipHostMallocDefault));
        char *cur_ptr = reads_CSR;
        T_CSR_cap *offs_ptr = reads_offs;
        *offs_ptr = 0;
        for (const ReadPtr &read_ptr: reads) {
            memcpy(cur_ptr, read_ptr.read, read_ptr.len);
            cur_ptr += read_ptr.len;
            offs_ptr++;
            *offs_ptr = *(offs_ptr-1) + read_ptr.len;
        }
    }
    PinnedCSR::~PinnedCSR() {
        CUDA_CHECK(hipHostFree(reads_offs));
        CUDA_CHECK(hipHostFree(reads_CSR));
    }


struct T_d_data {
    _in_ T_read_cnt reads_cnt;
    
    // Raw reads
    _in_ _out_ char *d_reads; // will be also used to store HPC reads (if HPC is enabled)
    _in_ T_CSR_cap *d_read_offs; // reads are in CSR format so offset array is required
    _in_ _out_ T_read_len *d_read_len;  // len == len(d_read_offs)  int
    
    // HPC reads info
    T_read_len *d_hpc_orig_pos;         // len == len(d_reads)      size_t  base original pos **in a read** (not in CSR)
    
    // Minimizers
    _out_ T_minimizer *d_minimizers;    // len == len(d_reads)      size_t
    _out_ T_kvalue *d_mm_pos;           // len == len(d_reads)      u_char  minimizer position in each window
    _out_ char *d_mm_strand;            // len == len(d_reads)      char    0 for forward, 1 for reverse complement, -1 for f==rc
    _out_ T_read_len *d_superkmer_offs; // len == len(d_reads)      int     supermer_offs **in a read**
}; // device data

__global__ void GPU_HPCEncoding (
    _in_ T_read_cnt d_reads_cnt, _out_ T_read_len *d_read_len, 
    _in_ _out_ char *d_reads, _in_ T_CSR_cap *d_read_offs, 
    bool HPC, _out_ T_read_len *d_hpc_orig_pos=NULL) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;
    if (!HPC) { // only calculate read_len in global memory (optional but essential for HPC=true)
        for (T_read_cnt rid = tid; rid < d_reads_cnt; rid += n_t) {
            d_read_len[rid] = d_read_offs[rid+1] - d_read_offs[rid];
        }
        __syncthreads();
        return;
    }
    
    for (T_read_cnt rid = tid; rid < d_reads_cnt; rid += n_t) {
        T_read_len read_len = d_read_offs[rid+1] - d_read_offs[rid];
        T_read_len last_idx = 0, hpc_arr_idx = d_read_offs[tid], j;
        d_hpc_orig_pos[hpc_arr_idx] = 0;
        for (T_read_len i = 1; i < read_len; i++) {
            j = i + d_read_offs[rid];
            last_idx += (i-last_idx) * (d_reads[j] != d_reads[j-1]);
            hpc_arr_idx += (d_reads[j] != d_reads[j-1]);
            d_hpc_orig_pos[hpc_arr_idx] = last_idx;
            d_reads[hpc_arr_idx] = d_reads[j];
        }
        d_read_len[rid] = hpc_arr_idx + 1 - d_read_offs[rid];
    }
    return;
}

// ======== Minimizer Functions ========
// traditional minimizer
__device__ __forceinline__ bool mm_filter(T_minimizer mm, int p) {
    // return mm%101>80; // 20.36
    // return ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/; // 19.94
    // return ((mm >> (p-2)*2) & 0b11) + ((mm >> (p-3)*2) & 0b11) + ((mm >> (p-1)*2) & 0b11); // 20.03
    // return (mm >> (p-3)*2) * ((mm >> (p-5)*2) & 0b111111); // 20.02
    // return ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/ & (mm >> ((p-3)*2) != 0b001000); // 19.92
    // int i=0;
    // int s=0;
    // for (i=1; i<3; i++) {
    //     s += (mm >> ((p-2)*2)) > (mm>>((p-2-i))&0b1111);
    // }
    // return s==0;
    return true;
}
// new design: 2nd/3rd不都为a
__device__ __forceinline__ bool new_filter(T_minimizer mm, int p) {
    return ((mm >> (p-2)*2) & 0b11) + ((mm >> (p-3)*2) & 0b11);
}
__device__ __forceinline__ bool new_filter2(T_minimizer mm, int p) {
    return ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/ & (mm >> ((p-3)*2) != 0b001000);
}
// KMC2 signature
__device__ bool sign_filter(T_minimizer mm, int p) {
    T_minimizer t = mm;
    bool flag = true;
    for (int ii = 0; ii < p-2; ii ++) {
        flag *= ((t & 0b1111) != 0);
        t = t >> 2;
    }
    // printf("%d Minimizer: %x\n", flag & ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100), mm);
    return flag & ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/;
}
/*
 * [INPUT]  data.reads in [(Read#0), (Read#1)...]
 * [OUTPUT] data.minimizers in [(Read#0)[mm1, mm?, mm?, ...], (Read#1)...]
 * all thread do one read at the same time with coalesced global memory access
 * TODO: make a 32-bit minimizer version and compare the performance
 */
__global__ void GPU_GenMinimizer( // TODO: reverse complement of minimizer
    _in_ T_read_cnt d_reads_cnt, _in_ T_read_len *d_read_len, 
    _in_ char *d_reads, _in_ T_CSR_cap *d_read_offs, 
    _out_ T_minimizer *d_minimizers, 
    int K_kmer, int P_minimizer) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;
    int i, j, cur_kmer_i;
    T_minimizer mm_mask = T_MM_MAX >> (sizeof(T_minimizer)*8 - 2*P_minimizer);
    T_minimizer mm_set; // selected minimizer
    T_minimizer mm, new_mm, mm_rc, new_mm_rc; // rc for reverse complement
    
    bool mm_check; // whether is a legal minimizer/signature (filtered by mm_filter)

    for (i=0; i<d_reads_cnt; i++) {
        char *read = &d_reads[d_read_offs[i]]; // current read
        T_minimizer *minimizer_saving = &(d_minimizers[d_read_offs[i]]);
        T_read_len len = d_read_len[i];
        for (cur_kmer_i=tid; cur_kmer_i <= len-K_kmer; cur_kmer_i+=n_t) { // Coalesced Access
            // gen the first p-mer:
            new_mm = 0;
            for (j = cur_kmer_i; j < cur_kmer_i + P_minimizer; j++) {
                new_mm = (new_mm << 2) | d_basemap[*(read+j)];
            }
            mm_check = FILTER_KERNEL(new_mm, P_minimizer);
            mm = new_mm * mm_check + mm_mask * (!mm_check); // if not a minimizer, let it be maximal (no minimizer can be maximal because canonical)
            
            // gen the first RC p-mer:
            new_mm_rc = 0;
            for (j = cur_kmer_i + P_minimizer - 1; j >= cur_kmer_i; j--) {
                new_mm_rc = (new_mm_rc << 2) | d_basemap_compl[*(read+j)];
            }
            mm_check = FILTER_KERNEL(new_mm_rc, P_minimizer);
            mm_rc = new_mm_rc * mm_check + mm_mask * (!mm_check);

            mm_set = (mm_rc < mm) * mm_rc + (mm_rc >= mm) * mm;////////////
            
            // gen the next p-mers:
            for (j = cur_kmer_i + P_minimizer; j < cur_kmer_i + K_kmer; j++) {
                // gen new minimizers
                new_mm = ((new_mm << 2) | d_basemap[*(read+j)]) & mm_mask;
                new_mm_rc = (new_mm_rc >> 2) | (d_basemap_compl[*(read+j)] << (P_minimizer*2-2));
                // check new minimizers
                mm_check = FILTER_KERNEL(new_mm, P_minimizer);
                mm = new_mm * mm_check + mm * (!mm_check);
                mm_check = FILTER_KERNEL(new_mm_rc, P_minimizer);
                mm_rc = new_mm_rc * mm_check + mm_rc * (!mm_check);
                // set the best minimizer
                mm_set = (mm_set < mm) * mm_set + (mm_set >= mm) * mm;
                mm_set = (mm_set < mm_rc) * mm_set + (mm_set >= mm_rc) * mm_rc;//////////
            }
            minimizer_saving[cur_kmer_i] = mm_set;
            // printf("mmset %x\n",mm_set);
        }
    }
    // if (tid==0) printf("--kernel end");
    return;
}


/* [INPUT]  data.minimizers in [[mm1, mm1, mm2, mm3, ...], ...]
 * [OUTPUT] data.superkmer_offs in [[0, 2, 3, ...], ...]
*/
__global__ void GPU_GenSKM(
    _in_ T_read_cnt d_reads_cnt, _in_ T_read_len *d_read_len, 
    _in_ T_CSR_cap *d_read_offs, 
    _in_ T_minimizer *d_minimizers,
    _out_ T_read_len *d_superkmer_offs,
    int K_kmer, int P_minimizer) {
        
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;

    for (T_read_cnt rid = tid; rid < d_reads_cnt; rid += n_t) {
        T_read_len len = d_read_len[rid];                               // current read length
        T_minimizer *minimizers = &(d_minimizers[d_read_offs[rid]]);   // minimizer list pointer
        T_read_len *skm = &d_superkmer_offs[d_read_offs[rid]];         // superkmer list pointer
        T_read_len last_skm_pos = 0, skm_count = 0;                      // position of the last minimizer; superkmer count
        skm[0] = 0;
        for (T_read_len i = 1; i <= len-K_kmer; i++) {
            skm_count += (minimizers[i] != minimizers[i-1]); // current minimizer != last minimizer, new skm generated
            last_skm_pos = (minimizers[i] == minimizers[i-1]) * last_skm_pos + (minimizers[i] != minimizers[i-1]) * i;
            skm[skm_count] = last_skm_pos;
        }
        // assert(len!=0);
        skm[skm_count+1] = len-K_kmer+1;
        skm[skm_count+2] = 0;
    }
    return;
}

/// @brief Set device CSR offsets begin from 0.
/// @param d_reads_cnt 
/// @param d_read_offs 
/// @param add [0] for setting to zero, [positive] value for adding back
/// @return
__global__ void MoveOffset(_in_ T_read_cnt d_reads_cnt, _in_ _out_ T_CSR_cap *d_read_offs, long long add=0) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;
    add = add - (add==0) * d_read_offs[0];
    for (T_read_cnt rid = tid; rid <= d_reads_cnt; rid += n_t) {
        d_read_offs[rid] += add;
    }
    return;
}


// post-GPU task functions
__host__ void SaveSKMs_instream (CountTask task) {
    if (task == CountTask::SKMPartition) {

    } else if (task == CountTask::SKMPartWithPos) {

    }
}
__host__ void CalcSKMPartSize_instream (T_read_cnt reads_cnt, T_read_len *superkmer_offs, 
    T_CSR_cap *reads_offs, T_minimizer *minimizers, 
    int n_partitions, int k, atomic<size_t> part_sizes[]) {
    int i;
    T_CSR_cap skm_offs_idx;
    T_CSR_cap offs_move = reads_offs[0];
    T_read_len superkmer_len;
    for (i=0; i<reads_cnt; i++) {
        skm_offs_idx = reads_offs[i] - offs_move + 1; // [1, n] 0~1 1~2 ... n-1~n
        while (superkmer_offs[skm_offs_idx] != 0) {
            superkmer_len = superkmer_offs[skm_offs_idx]-1 - superkmer_offs[skm_offs_idx-1] + k;
            part_sizes[minimizers[superkmer_offs[skm_offs_idx-1] + reads_offs[i] - offs_move] % n_partitions] += superkmer_len;
            skm_offs_idx++;
        }
    }
}

__host__ void GPUReset(int did) {
    // do not call it after host malloc
    CUDA_CHECK(hipSetDevice(did));
    CUDA_CHECK(hipDeviceReset());
    CUDA_CHECK(hipDeviceSynchronize());
    return;
}

// provide pinned_reads from the shortest to the longest read
__host__ void GenSuperkmerGPU (PinnedCSR &pinned_reads, 
    int K_kmer, int P_minimizer, bool HPC, CUDAParams gpars, CountTask task,
    int SKM_partitions, std::function<void(T_h_data)> process_func
    /*atomic<size_t> skm_part_sizes[]*/) {
    
    int time_all=0, time_filter=0;

    CUDA_CHECK(hipSetDevice(gpars.device_id));
    CUDA_CHECK(hipDeviceSynchronize());
    
    hipStream_t streams[gpars.n_streams];
    T_d_data gpu_data[gpars.n_streams];
    T_h_data host_data[gpars.n_streams];
    T_CSR_cap batch_size[gpars.n_streams];
    T_read_cnt bat_beg_read[gpars.n_streams];//, bat_end_read[gpars.n_streams];

    int i, started_streams;
    for (i=0; i<gpars.n_streams; i++)
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    
    T_read_cnt items_per_stream = gpars.NUM_BLOCKS_PER_GRID * gpars.NUM_THREADS_PER_BLOCK;
    T_read_cnt cur_read = 0, end_read;
    i = 0; // i for which stream
    while (cur_read < pinned_reads.n_reads) {

        // TODO: check if last round is finished if CPU postprocess func is async

        for (i = 0; i < gpars.n_streams && cur_read < pinned_reads.n_reads; i++, cur_read += items_per_stream) {
            bat_beg_read[i] = cur_read;
            end_read = min(cur_read + items_per_stream, pinned_reads.n_reads); // the last read in this stream batch
            // bat_end_read[i] = end_read;
            host_data[i].reads_cnt = gpu_data[i].reads_cnt = end_read-cur_read;
            batch_size[i] = pinned_reads.reads_offs[end_read] - pinned_reads.reads_offs[cur_read]; // read size in bytes
            // gpu_data[i].offs_move = pinned_reads.reads_offs[cur_read];
            logger->log("GPU stream "+to_string(i)+":\tread count = "+to_string(gpu_data[i].reads_cnt));

            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            // 1. hipMalloc (5000 reads / GB)
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_reads), sizeof(char) * (batch_size[i]+1), streams[i]));//                // 8192 threads(reads) * 20 KB/read     = 160MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_read_offs), sizeof(T_CSR_cap) * (gpu_data[i].reads_cnt+1), streams[i])); // 8192 threads(reads) * 8 B/read       =  64MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_read_len), sizeof(T_read_len) * (gpu_data[i].reads_cnt), streams[i]));   // 8192 threads(reads) * 4 B/read       =  32MB VRAM
            if (HPC) {// cost a lot VRAM
                CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_hpc_orig_pos), sizeof(T_read_len) * (batch_size[i]), streams[i]));   // 8192 threads(reads) * 20K * 4B/read  = 640MB VRAM
            } else {
                gpu_data[i].d_hpc_orig_pos = NULL;
            }
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_minimizers), sizeof(T_minimizer) * (batch_size[i]), streams[i]));        // 8192 threads(reads) * 20K * 4B/read  = 640MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_superkmer_offs), sizeof(T_read_len) * (batch_size[i]), streams[i]));     // 8192 threads(reads) * 20K * 4B/read  = 640MB VRAM
            
            // 2. hipMemcpy (H2D) // TODO: async
            CUDA_CHECK(hipMemcpyAsync(gpu_data[i].d_reads, &(pinned_reads.reads_CSR[pinned_reads.reads_offs[cur_read]]), batch_size[i], hipMemcpyHostToDevice, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(gpu_data[i].d_read_offs, &(pinned_reads.reads_offs[cur_read]), sizeof(T_CSR_cap) * (gpu_data[i].reads_cnt+1), hipMemcpyHostToDevice, streams[i]));
            
            // 3. GPU Computing
            #ifdef KERNEL_TIME_MEASUREMENT
            WallClockTimer wct;
            #endif
            MoveOffset<<<gpars.NUM_BLOCKS_PER_GRID, gpars.NUM_THREADS_PER_BLOCK, 0, streams[i]>>>(
                gpu_data[i].reads_cnt, gpu_data[i].d_read_offs, 0
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            #endif
            
            GPU_HPCEncoding<<<gpars.NUM_BLOCKS_PER_GRID, gpars.NUM_THREADS_PER_BLOCK, 0, streams[i]>>>(
                gpu_data[i].reads_cnt, gpu_data[i].d_read_len, 
                gpu_data[i].d_reads, gpu_data[i].d_read_offs, 
                HPC, gpu_data[i].d_hpc_orig_pos
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            
            WallClockTimer wct2;
            #endif
            // hipEventRecord(start, streams[i]);
            GPU_GenMinimizer<<<gpars.NUM_BLOCKS_PER_GRID, gpars.NUM_THREADS_PER_BLOCK, 0, streams[i]>>>(
                gpu_data[i].reads_cnt, gpu_data[i].d_read_len,
                gpu_data[i].d_reads, gpu_data[i].d_read_offs,
                gpu_data[i].d_minimizers, 
                K_kmer, P_minimizer
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            time_filter += wct2.stop(true);
            #endif
            // hipEventRecord(stop, streams[i]);
            // hipEventElapsedTime(&time_tmp, start, stop); time_all += time_tmp; time_filter += time_tmp;

            GPU_GenSKM<<<gpars.NUM_BLOCKS_PER_GRID, gpars.NUM_THREADS_PER_BLOCK, 0, streams[i]>>>(
                gpu_data[i].reads_cnt, gpu_data[i].d_read_len,
                gpu_data[i].d_read_offs, 
                gpu_data[i].d_minimizers,
                gpu_data[i].d_superkmer_offs,
                K_kmer, P_minimizer
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            time_all += wct.stop(true);
            #endif
            // pinned_reads.reads_offs[cur_read]
        }
        started_streams = i;
        for (i = 0; i < started_streams; i++) {
            // -- Malloc on host for temporary result storage --
            // TODO: add if on task to indicate whether to new and D2H
            if (HPC) {
                host_data[i].hpc_orig_pos = new T_read_len[batch_size[i]];
                host_data[i].read_len = new T_read_len[gpu_data[i].reads_cnt];
            }
            host_data[i].minimizers = new T_minimizer[batch_size[i]];
            host_data[i].reads = &(pinned_reads.reads_CSR[pinned_reads.reads_offs[bat_beg_read[i]]]); // used pinned memory to store the output
            host_data[i].reads_offs = &(pinned_reads.reads_offs[bat_beg_read[i]]); // !!! offs not begin from 0. used pinned memory to store the output
            host_data[i].superkmer_offs = new T_read_len[batch_size[i]];

            // -- D2H memory copy --
            if (HPC) {
                CUDA_CHECK(hipMemcpyAsync(host_data[i].hpc_orig_pos, gpu_data[i].d_hpc_orig_pos, sizeof(T_read_len) * batch_size[i], hipMemcpyDeviceToHost, streams[i]));
                CUDA_CHECK(hipMemcpyAsync(host_data[i].read_len, gpu_data[i].d_read_len, sizeof(T_read_len) * host_data[i].reads_cnt, hipMemcpyDeviceToHost, streams[i]));
            }
            CUDA_CHECK(hipMemcpyAsync(host_data[i].minimizers, gpu_data[i].d_minimizers, sizeof(T_minimizer) * batch_size[i], hipMemcpyDeviceToHost, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(host_data[i].reads, gpu_data[i].d_reads, sizeof(char) * batch_size[i], hipMemcpyDeviceToHost, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(host_data[i].superkmer_offs, gpu_data[i].d_superkmer_offs, sizeof(T_read_len) * batch_size[i], hipMemcpyDeviceToHost, streams[i]));

            // -- Free device memory --
            if (HPC) CUDA_CHECK(hipFreeAsync(gpu_data[i].d_hpc_orig_pos, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_minimizers, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_read_len, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_reads, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_read_offs, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_superkmer_offs, streams[i]));
            
            // -- Wait for CUDA --
            CUDA_CHECK(hipStreamSynchronize(streams[i])); // move this into async post_proc_func
            // logger->log("GPU Done"+to_string(i));
            
            // -- CPU post-process -- // TODO: async & use bind to pass the post process function

            // future<void> file_loading_res = async(std::launch::async, process_func);
            process_func(host_data[i]);
            // CalcSKMPartSize_instream(host_data[i].reads_cnt, host_data[i].superkmer_offs, host_data[i].reads_offs, host_data[i].minimizers, SKM_partitions, K_kmer, skm_part_sizes);
            
            // -- clean host variables --
            // (TODO: 如果post-process 用 async, 则free放在post-process函数里，且保证host_data非引用传递给async proc func以防下一轮更新)
            if (HPC) {
                delete [] host_data[i].hpc_orig_pos;
                delete [] host_data[i].read_len;
            }
            delete [] host_data[i].minimizers;
            delete [] host_data[i].superkmer_offs;
        }
    }
    logger->log("FILTER: " STR(FILTER_KERNEL) " Kernel Functions Time: ALL = "+to_string(time_all)+"ms FILTER = "+to_string(time_filter)+"ms");
}