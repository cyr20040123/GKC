#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
// #include <thrust/scan.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>

#include "types.h"
#include "superkmers.hpp"
#include <vector>
using namespace std;

// struct differentfromprev {
//     differentfromprev() {}
//     __host__ __device__
//         bool operator()(const T_kmer& x, const T_kmer& y) const { 
//             return x!=y;
//         }
// };
struct sameasprev {
    sameasprev() {}
    __host__ __device__
        bool operator()(const T_kmer& x, const T_kmer& y) const { 
            return x==y;
        }
};
struct canonicalkmer {
    canonicalkmer() {}
    __host__ __device__
        T_kmer operator()(const T_kmer& x, const T_kvalue k) const {
            T_kmer x1 = ~x, res=0;
            for (T_kvalue i=0; i<k; i++) {
                res = (res << 2) | (x1 & 0b11);
                x1 = x1 >> 2;
            }
            return res < x ? res : x;
        }
};
struct replaceidx {
    replaceidx() {}
    __host__ __device__
        T_read_len operator()(const T_read_len& x, const T_read_len& y) const {
            return x*y;
        }
};
struct is_zero {
    __host__ __device__
        bool operator()(const T_read_len x)
        {
            return x==0;
        }
};
__host__ size_t kmc_counting_GPU (T_kvalue k,
                               SKMStore &skms_store, int gpuid,
                               unsigned short kmer_min_freq, unsigned short kmer_max_freq,
                               _out_ vector<T_kmc> &kmc_result_curthread) {
    // using CUDA Thrust
    // size_t est_kmer = skms_store.tot_size - skms_store.skms.size_approx() * (k-1);
    // size_t db_skm = skms_store.skms.size_approx();
    // size_t est_skm = 0;
    
    // 0. Extract kmers from SKMStore:
    if (skms_store.tot_size == 0) return 0;
    thrust::host_vector<T_kmer> kmers_h;
    size_t tot_kmers;
    T_skm_len skm_len;
    T_kmer kmer_mask = T_kmer(0xffffffffffffffff>>(64-k*2));
    T_kmer kmer;
    CRead<T_skm_len> skm;
    while (skms_store.try_pop_skm(skm)) {
        skm_len = skm.length();
        // first kmer
        kmer = skm.get_2bit_kmer(0, k);
        kmers_h.push_back(kmer);
        // latter kmers
        for (T_skm_len j=k; j<skm_len; j++) {
            kmer = ((kmer << 2) | skm.get_2bit(j)) & kmer_mask;
            kmers_h.push_back(kmer);
        }
    }
    thrust::device_vector<T_kmer> kmers_d(kmers_h);
    tot_kmers = kmers_d.size();
    // cerr<<est_kmer<<"|"<<db_skm<<"|"<<est_skm<<"|"<<tot_kmers<<endl;
    
    // 1. convert to canonical kmers
    thrust::constant_iterator<T_kvalue> ik(k);
    thrust::transform(thrust::device, kmers_d.begin(), kmers_d.end(), ik, kmers_d.begin(), canonicalkmer());

    // 2. sort: [ABCBBAC] -> [AABBBCC] (kmers_d)
    thrust::sort(thrust::device, kmers_d.begin(), kmers_d.end()/*, thrust::greater<T_kmer>()*/);
    thrust::host_vector<T_kmer> sorted_kmers_h = kmers_d;

    // 3. find changes: [AABBBCC] -> [0,0,1,0,0,1,0] (comp_vec_d)
    // thrust::device_vector<bool> comp_vec_d(kmers_d.size());
    // thrust::transform(thrust::device, kmers_d.begin()+1 /*x beg*/, kmers_d.end() /*x end*/, kmers_d.begin()/*y beg*/, comp_vec_d.begin()+1/*res beg*/, differentfromprev());
    // comp_vec_d[0] = 1; //
    // int distinct_kmer_cnt = thrust::reduce(thrust::device, comp_vec_d.begin(), comp_vec_d.end()) + 1;
    
    // 3. find changes: [AABBBCC] -> [0,1,0,1,1,0,1] (same_flag_d)
    thrust::device_vector<bool> same_flag_d(kmers_d.size());
    thrust::transform(thrust::device, kmers_d.begin()+1 /*x beg*/, kmers_d.end() /*x end*/, kmers_d.begin()/*y beg*/, same_flag_d.begin()+1/*res beg*/, sameasprev());
    same_flag_d[0] = 0; //
    
    // 4. remove same idx: [0123456] [0101101] -> [0,2,5] (idx_d)
    thrust::device_vector<T_read_len> idx_d(kmers_d.size());
    thrust::sequence(thrust::device, idx_d.begin(), idx_d.end());
    auto new_end_d = thrust::remove_if(thrust::device, idx_d.begin(), idx_d.end(), same_flag_d.begin(), thrust::identity<bool>()); // new_end_d is an iterator

    // 4. replace with index: [0,0,1,0,0,1,0] -> [0,0,2,0,0,5,0] (comp_vec_d)
    // thrust::device_vector<T_read_len> seq_d(kmers_d.size());
    // thrust::sequence(thrust::device, seq_d.begin(), seq_d.end());
    // thrust::transform(thrust::device, comp_vec_d.begin() /*x*/, comp_vec_d.end(), seq_d.begin()/*y*/, comp_vec_d.begin()/*res*/, replaceidx());

    // // 5. skip repeats: [0,0,2,0,0,5,0] -> [0,2,5] (comp_vec_d)
    // auto new_end_d = thrust::remove_if(thrust::device, comp_vec_d.begin(), comp_vec_d.end(), is_zero());

    // 5. copy device_vector back to host_vector
    thrust::host_vector<T_read_len> idx_h(idx_d.begin(), new_end_d);
    idx_h.push_back(tot_kmers); // [0,2,5] -> [0,2,5,7] A2 B3 C2
    
    size_t total_kmer_cnt = 0;
    int i;
    T_kmer_cnt cnt;
    for(i=0; i<idx_h.size()-1; i++) {
        cnt = idx_h[i+1]-idx_h[i] > MAX_KMER_CNT ? MAX_KMER_CNT : idx_h[i+1]-idx_h[i];
        total_kmer_cnt += idx_h[i+1]-idx_h[i];
        // Add kmer-cnt to result vector:
        // if (cnt >= kmer_min_freq && cnt <= kmer_max_freq) {
        //     kmc_result_curthread.push_back({sorted_kmers_h[idx_h[i]], cnt});
        // }
    }
    // return total_kmer_cnt; // total kmer
    return idx_h.size()-1; // total distinct kmer
}

// kmc_counting_GPU (skms, gpuid, kmer_min_freq, kmer_max_freq, kmc_result_curthread);